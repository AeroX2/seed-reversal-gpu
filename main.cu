
// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif




#include <hip/hip_runtime.h>
#include <memory.h>
#include <stdio.h>


#define signed_seed_t long long
#define uint unsigned int
#define ulong unsigned signed_seed_t

#undef JRAND_DOUBLE

#define RANDOM_MULTIPLIER_LONG 0x5DEECE66DUL

#ifdef JRAND_DOUBLE
#define Random double
#define RANDOM_MULTIPLIER 0x5DEECE66Dp-48
#define RANDOM_ADDEND 0xBp-48
#define RANDOM_SCALE 0x1p-48

inline uint random_next(Random *random, int bits) {
  *random = trunc((*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) * RANDOM_SCALE);
  return (uint)((ulong)(*random / RANDOM_SCALE) >> (48 - bits));
}

#else

#define Random ulong
#define RANDOM_MULTIPLIER RANDOM_MULTIPLIER_LONG
#define RANDOM_ADDEND 0xBUL
#define RANDOM_MASK (1UL << 48) - 1
#define RANDOM_SCALE 1

#define FAST_NEXT_INT

// Random::next(bits)
__host__ __device__ inline uint random_next(Random *random, int bits) {
    *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
    return (uint)(*random >> (48 - bits));
}
#endif // ~JRAND_DOUBLE

// new Random(seed)
#define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER_LONG) & RANDOM_MASK))
#define get_random_unseeded(state) ((Random) ((state) * RANDOM_SCALE))

// Random::nextInt(bound)
__host__ __device__ inline uint random_next_int(Random *random, uint bound) {
    int r = random_next(random, 31);
    int m = bound - 1;
    if ((bound & m) == 0) {
        r = (uint)((bound * (ulong)r) >> 31);
    } else {
#ifdef FAST_NEXT_INT
        r %= bound;
#else
        for (int u = r;
             u - (r = u % bound) + m < 0;
             u = random_next(random, 31));
#endif
    }
    return r;
}

__host__ __device__ inline long random_next_long (Random *random) {
    return (((long)random_next(random, 32)) << 32) + random_next(random, 32);
}

// advance
#define advance(rand, multiplier, addend) ((rand) = ((rand) * (multiplier) + (addend)) & RANDOM_MASK)
#define advance_830(rand) advance(rand, 0x859D39E832D9LL, 0xE3E2DF5E9196LL)
#define advance_774(rand) advance(rand, 0xF8D900133F9LL, 0x5738CAC2F85ELL)
#define advance_387(rand) advance(rand, 0x5FE2BCEF32B5LL, 0xB072B3BF0CBDLL)
#define advance_16(rand) advance(rand, 0x6DC260740241LL, 0xD0352014D90LL)
#define advance_m1(rand) advance(rand, 0xDFE05BCB1365LL, 0x615C0E462AA9LL)
#define advance_m3759(rand) advance(rand, 0x63A9985BE4ADLL, 0xA9AA8DA9BC9BLL)



#define TREE_X 4
#define TREE_Z 3
#define TREE_HEIGHT 6

#define OTHER_TREE_COUNT 3
__device__ inline int getTreeHeight(int x, int z) {
    if (x == TREE_X && z == TREE_Z)
        return TREE_HEIGHT;

    if (x == 1 && z == 13)
        return 5;

    if (x == 6 && z == 12)
        return 6;

    if (x == 14 && z == 7) {
        return 5;
    }

    return 0;
}

#define WATERFALL_X 9
#define WATERFALL_Y 76
#define WATERFALL_Z 1



#define MODULUS (1LL << 48)
#define X_TRANSLATE 0
#define Z_TRANSLATE 11
#define L00 7847617LL
#define L01 (-18218081LL)
#define L10 4824621LL
#define L11 24667315LL
#define LI00 (24667315.0 / 16)
#define LI01 (18218081.0 / 16)
#define LI10 (-4824621.0 / 16)
#define LI11 (7847617.0 / 16)

#define CONST_FLOOR(x) ((x) < (signed_seed_t) (x) ? (signed_seed_t) (x) - 1 : (signed_seed_t) (x))
#define CONST_CEIL(x) ((x) == (signed_seed_t) (x) ? (signed_seed_t) (x) : CONST_FLOOR((x) + 1))
#define CONST_LOWER(x, m, c) ((m) < 0 ? ((x) + 1 - (double) (c) / MODULUS) * (m) : ((x) - (double) (c) / MODULUS) * (m))
#define CONST_UPPER(x, m, c) ((m) < 0 ? ((x) - (double) (c) / MODULUS) * (m) : ((x) + 1 - (double) (c) / MODULUS) * (m))

#define LOWER_X CONST_FLOOR(CONST_LOWER(TREE_X, LI00, X_TRANSLATE) + CONST_LOWER(TREE_Z, LI01, Z_TRANSLATE))
#define LOWER_Z CONST_FLOOR(CONST_LOWER(TREE_X, LI10, X_TRANSLATE) + CONST_LOWER(TREE_Z, LI11, Z_TRANSLATE))
#define UPPER_X CONST_CEIL(CONST_UPPER(TREE_X, LI00, X_TRANSLATE) + CONST_UPPER(TREE_Z, LI01, Z_TRANSLATE))
#define UPPER_Z CONST_CEIL(CONST_UPPER(TREE_X, LI10, X_TRANSLATE) + CONST_UPPER(TREE_Z, LI11, Z_TRANSLATE))
#define SIZE_X (UPPER_X - LOWER_X + 1)
#define SIZE_Z (UPPER_Z - LOWER_Z + 1)
#define TOTAL_WORK_SIZE (SIZE_X * SIZE_Z)

#define MAX_TREE_ATTEMPTS 12
#define MAX_TREE_SEARCH_BACK (3 * MAX_TREE_ATTEMPTS - 3 + 16 * OTHER_TREE_COUNT)

#define WORK_UNIT_SIZE (1LL << 20)
#define BLOCK_SIZE 256



__global__ void doWork(ulong offset, int* num_seeds, ulong* seeds) {
    // lattice tree position
    ulong global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset + global_id >= TOTAL_WORK_SIZE) return;

    signed_seed_t lattice_x = (signed_seed_t) ((offset + global_id) % SIZE_X) + LOWER_X;
    signed_seed_t lattice_z = (signed_seed_t) ((offset + global_id) / SIZE_X) + LOWER_Z;
    Random rand = (Random) ((lattice_x * L00 + lattice_z * L01 + X_TRANSLATE) % MODULUS);
    advance_m1(rand);
    Random start = rand;
    advance_m1(start);

    bool res = random_next(&rand, 4) == TREE_X;
    res &= random_next(&rand, 4) == TREE_Z;
    res &= random_next_int(&rand, 3) == (ulong) (TREE_HEIGHT - 4);


    for (int treeBackCalls = 0; treeBackCalls <= MAX_TREE_SEARCH_BACK; treeBackCalls++) {
        rand = start;

        bool this_res = res;
        this_res &= random_next_int(&rand, 10) != 0;

        bool generated_tree[16][16];
        memset(generated_tree, false, sizeof(generated_tree));

        int treesMatched = 0;
        bool any_population_matches = false;
        for (int treeAttempt = 0; treeAttempt <= MAX_TREE_ATTEMPTS; treeAttempt++) {
            int treeX = random_next(&rand, 4);
            int treeZ = random_next(&rand, 4);
            int wantedTreeHeight = getTreeHeight(treeX, treeZ);
            int treeHeight = random_next_int(&rand, 3) + 4;
            if (treeHeight == wantedTreeHeight && !generated_tree[treeX][treeZ]) {
                treesMatched++;
                generated_tree[treeX][treeZ] = true;
                advance_16(rand);
            }

            if (treesMatched == OTHER_TREE_COUNT + 1) {
                Random before_rest = rand;
                // yellow flowers
                advance_774(rand);
                // red flowers
                if (random_next(&rand, 1) == 0) {
                    advance_387(rand);
                }
                // brown mushroom
                if (random_next(&rand, 2) == 0) {
                    advance_387(rand);
                }
                // red mushroom
                if (random_next(&rand, 3) == 0) {
                    advance_387(rand);
                }
                // reeds
                advance_830(rand);
                // pumpkins
                if (random_next(&rand, 5) == 0) {
                    advance_387(rand);
                }

                for (int i = 0; i < 50; i++) {
                    bool waterfall_matches = random_next(&rand, 4) == WATERFALL_X;
                    waterfall_matches &= random_next_int(&rand, random_next_int(&rand, 120) + 8) == WATERFALL_Y;
                    waterfall_matches &= random_next(&rand, 4) == WATERFALL_Z;
                    any_population_matches |= waterfall_matches;
                }
                rand = before_rest;
            }
        }

        this_res &= any_population_matches;

        Random start_chunk_rand = start;
        advance_m3759(start_chunk_rand);
        if (this_res) {
            int index = atomicAdd(num_seeds, 1);
            seeds[index] = start_chunk_rand;
        }

        advance_m1(start);
    }

}

int main() {
    printf("Searching %lld total seeds...\n", TOTAL_WORK_SIZE);

    FILE* out_file = fopen("chunk_seeds.txt", "w");


    int* num_seeds;
    hipMallocManaged(&num_seeds, sizeof(*num_seeds));

    ulong* seeds;
    hipMallocManaged(&seeds, (1LL << 30)); // approx 1gb

    ulong count = 0;
    for (ulong offset = 0; offset < TOTAL_WORK_SIZE; offset += WORK_UNIT_SIZE) {
        *num_seeds = 0;

        doWork <<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (offset, num_seeds, seeds);
        hipDeviceSynchronize();

        for (int i = 0, e = *num_seeds; i < e; i++) {
            fprintf(out_file, "%lld\n", seeds[i]);
        }
        fflush(out_file);

        count += *num_seeds;
        printf("Searched %lld seeds, found %lld matches \n", offset + WORK_UNIT_SIZE, count);
    }

    fclose(out_file);

}