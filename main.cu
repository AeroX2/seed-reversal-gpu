
// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif




#include <hip/hip_runtime.h>
#include <stdio.h>


#define signed_seed_t long long
#define uint unsigned int
#define ulong unsigned signed_seed_t

#undef JRAND_DOUBLE

#define RANDOM_MULTIPLIER_LONG 0x5DEECE66DUL

#ifdef JRAND_DOUBLE
#define Random double
#define RANDOM_MULTIPLIER 0x5DEECE66Dp-48
#define RANDOM_ADDEND 0xBp-48
#define RANDOM_SCALE 0x1p-48

inline uint random_next(Random *random, int bits) {
  *random = trunc((*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) * RANDOM_SCALE);
  return (uint)((ulong)(*random / RANDOM_SCALE) >> (48 - bits));
}

#else

#define Random ulong
#define RANDOM_MULTIPLIER RANDOM_MULTIPLIER_LONG
#define RANDOM_ADDEND 0xBUL
#define RANDOM_MASK (1UL << 48) - 1
#define RANDOM_SCALE 1

#define FAST_NEXT_INT

// Random::next(bits)
__host__ __device__ inline uint random_next(Random *random, int bits) {
    *random = (*random * RANDOM_MULTIPLIER + RANDOM_ADDEND) & RANDOM_MASK;
    return (uint)(*random >> (48 - bits));
}
#endif // ~JRAND_DOUBLE

// new Random(seed)
#define get_random(seed) ((Random)((seed ^ RANDOM_MULTIPLIER_LONG) & RANDOM_MASK))
#define get_random_unseeded(state) ((Random) ((state) * RANDOM_SCALE))

// Random::nextInt(bound)
__host__ __device__ inline uint random_next_int(Random *random, uint bound) {
    int r = random_next(random, 31);
    int m = bound - 1;
    if ((bound & m) == 0) {
        r = (uint)((bound * (ulong)r) >> 31);
    } else {
#ifdef FAST_NEXT_INT
        r %= bound;
#else
        for (int u = r;
             u - (r = u % bound) + m < 0;
             u = random_next(random, 31));
#endif
    }
    return r;
}

__host__ __device__ inline long random_next_long (Random *random) {
    return (((long)random_next(random, 32)) << 32) + random_next(random, 32);
}

// advance
#define advance_m1(rand) ((rand) = ((rand) * 0xDFE05BCB1365LL + 0x615C0E462AA9LL) & RANDOM_MASK)



#define TREE_X 0
#define TREE_Z 0
#define TREE_HEIGHT 3

#define OTHER_TREE_COUNT 1
__constant__ const int OTHER_TREE_XS[OTHER_TREE_COUNT] = { 1 };



#define MODULUS (1LL << 48)
#define X_TRANSLATE 0
#define Z_TRANSLATE 11
#define L00 7847617LL
#define L01 4824621LL
#define L10 (-18218081LL)
#define L11 24667315LL
#define LI00 (24667315.0 / 16)
#define LI01 (-4824621.0 / 16)
#define LI10 (18218081.0 / 16)
#define LI11 (7847617.0 / 16)

#define CONST_FLOOR(x) ((x) < (signed_seed_t) (x) ? (signed_seed_t) (x) - 1 : (signed_seed_t) (x))
#define CONST_CEIL(x) ((x) == (signed_seed_t) (x) ? (signed_seed_t) (x) : CONST_FLOOR((x) + 1))
#define CONST_LOWER(x, m, c) ((m) < 0 ? ((x) + 1 - (double) (c) / MODULUS) * (m) : ((x) - (double) (c) / MODULUS) * (m))
#define CONST_UPPER(x, m, c) ((m) < 0 ? ((x) - (double) (c) / MODULUS) * (m) : ((x) + 1 - (double) (c) / MODULUS) * (m))

#define LOWER_X CONST_FLOOR(CONST_LOWER(TREE_X, LI00, X_TRANSLATE) + CONST_LOWER(TREE_Z, LI01, Z_TRANSLATE))
#define LOWER_Z CONST_FLOOR(CONST_LOWER(TREE_X, LI10, X_TRANSLATE) + CONST_LOWER(TREE_Z, LI11, Z_TRANSLATE))
#define UPPER_X CONST_CEIL(CONST_UPPER(TREE_X, LI00, X_TRANSLATE) + CONST_UPPER(TREE_Z, LI01, Z_TRANSLATE))
#define UPPER_Z CONST_CEIL(CONST_UPPER(TREE_X, LI10, X_TRANSLATE) + CONST_UPPER(TREE_Z, LI11, Z_TRANSLATE))
#define SIZE_X (UPPER_X - LOWER_X + 1)
#define SIZE_Z (UPPER_Z - LOWER_Z + 1)
#define TOTAL_WORK_SIZE (SIZE_X * SIZE_Z)

#define WORK_UNIT_SIZE (1LL << 32)
#define BLOCK_SIZE 256



__global__ void map(ulong offset, bool* result) {
    // lattice tree position
    ulong global_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset + global_id >= TOTAL_WORK_SIZE) return;

    signed_seed_t lattice_x = (signed_seed_t) ((offset + global_id) % SIZE_X) + LOWER_X;
    signed_seed_t lattice_z = (signed_seed_t) ((offset + global_id) / SIZE_X) + LOWER_Z;
    Random rand = (Random) ((lattice_x * L00 + lattice_z * L10 + X_TRANSLATE) % MODULUS);

    advance_m1(rand);
    Random start = rand;

    bool res = random_next(&rand, 4) == 0;
    res &= random_next(&rand, 4) == 0;
    res &= random_next_int(&rand, 3) == (ulong) (TREE_HEIGHT - 4);



    result[global_id] = res;

}

int main() {
    printf("%f\n", LI01);
    printf("[%lld, %lld, %lld, %lld]: %lld * %lld = %lld\n", LOWER_X, LOWER_Z, UPPER_X, UPPER_Z, SIZE_X, SIZE_Z, TOTAL_WORK_SIZE);


    bool* result;
    hipMallocManaged(&result, WORK_UNIT_SIZE);


    ulong count = 0;
    for (ulong offset = 0; offset < TOTAL_WORK_SIZE; offset += WORK_UNIT_SIZE) {
        map <<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>> (offset, result);
        hipDeviceSynchronize();

        for (ulong i = 0; i < WORK_UNIT_SIZE; i++) {
            if (result[i])
                count++;
        }
        printf("%lld\n", count);
    }

}